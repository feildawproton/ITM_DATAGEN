#include "hip/hip_runtime.h"
#include "math.h"

__host__ __device__ __forceinline__
void raytrace_maxheight(const uint64_t const uint64_t source_x, const uint64_t source_y, const uint64_t dest_x, const uint64_t dest_y, float** height_map)
{
	const uint64_t x_signed_len = dest_x - source_x;
	const uint64_t y_signed_len = dest_y - source_y;
	const uint64_t length = max(abs(x_signed_len), abs(y_signed_len));
	float max_height = -9999.0;
	for(uint64_t offset = 0; offset < (length + 1); offset++)			//length + 1 to include endpoint
	{
		float fraction = ((float)offset) / ((float)length);
	}
}


