#include "hip/hip_runtime.h"
#include "calc_loss_funcs.cuh"
#include <math.h>

//this is based off of Nicole Patterson's right up on Signal Propagation Equations for ITM
//this function (insert here) takes arrays a parameters, each index representing a single examples
//and returns a array of power losses, each eantry being the result for a single example

//this functions calculates a vaiable h
//as far as I can tell this is alse C_obs
__device__ float calc_h(const float h_0, const float h_1, const float h_2, const float d_1, const float d_2)
{
	//h_ER is the height of the surface curvature at the obstruction point in meters
	float h_ER = (d_1 * d_2) / 16.944;
	float h = h_0 + h_ER - h_1 - ((h_2 - h_1) / (d_1 + d_2))*d_1;
	return h;
}

//lambda is the wavelenght
__device__ float calc_lambda(float freq)
{
	return (299792458.0 / freq);
}


//v is the geometry factor
__device__ float calc_v(const float h_0, const float h_1, const float h_2, const float d_1, const float d_2, const float freq)
{
	//lambda is the wavelenght
	float lam = calc_lambda(freq);
	float h = calc_h(h_0, h_1, h_2, d_1, d_2);
	
	//v is the geometry factor
	//using sqrtf to ensure float version.  even though nvcc will perform it's own insertion
	float v = h * sqrtf((2.0*(d_1 + d_2)) / (lam * d_1 * d_2));
	return v;
}

//R_FR is 60% of the first Fresnel Zone radius
__device__ float calc_R_FR(const float d_1, const float d_2, const float freq)
{
	float f_MHz = freq / 1000000.0;
	//using sqrtf to ensure float version.  even though nvcc will perform it's own insertion
	float R_FR = 0.6*(547.533*sqrtf((d_1*d_2) / (f_MHz*(d_1 + d_2) ) ) );
	return R_FR;
}

//this is the loss for a single example
//h_0 is the height of the obstruction in METERS
//h_1 is the height of the transmitter in METERS
//h_2 is the height of the receiver in METERS
//d_1 is the distance from the transmitter the obstruction point in KILOMETERS
//d_2 is the distance from the obstruction to the receiver in KILOMETERS
//freq is the frequency in Hertz (1/s)
__device__ float calc_loss(const float h_0, const float h_1, const float h_2, const float d_1, const float d_2, const float freq)
{
	float loss = 0.0;
	
	//v is the geometry factor
	float v = calc_v(h_0, h_1, h_2, d_1, d_2, freq);
	
	//accumulate loss from these various factors
	//FSPL loss occurs in the Fresnel Zone
	if(v <= -1.0)
	{
		//assumig the base is 10
		//using the float version instead of the default double version
		//hopefully nvcc makes the appropriate replacements
		float f_GHz = freq / 1000000000.0;
		loss += 20.0 * log10f(d_1 + d_2) + 20.0 * log10f(f_GHz) + 92.45;
	}
	//LOS loss occurs when the Freznel Zone is obstructed but the LOS line remains unobstructed
	if(v > 0.0 && v < 1.0)
	{
		//C_obs is the distance betweent he LOS and the obstruction
		float C_obs = calc_h(h_0, h_1, h_2, d_1, d_2);
		//R_FR is 60% of the first Fresnel Zone radius
		float R_FR = calc_R_FR(d_1, d_2, freq);
		loss += 6.0*(1.0 - (C_obs / R_FR));
	}
	//NLOS occurs whe the LOS is obstructed
	if(v >= 0.0)
	{
		//using log10f base 10
		//using float version of both log and sqrt
		loss += 6.9 + 20.0*log10f(sqrtf((v-0.1)*(v-0.1) + 1.0) + v - 0.1);
	}
	return loss;	
}
