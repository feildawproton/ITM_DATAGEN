#include "hip/hip_runtime.h"
#include "raytrace_maxheight.cuh"
#include "calc_loss_funcs.cuh"
#include <math.h>

__global__ void calc_losses(
	const int source_x, const int source_y, const int source_z, 
	const float *height_map, float *pLoss, const int y_size, const int x_size, 
	const unsigned y_threads, const unsigned x_threads, const float freq)
{
	int h_1 = source_z;
	
	unsigned gind_y = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned gind_x = threadIdx.x + blockIdx.x * blockDim.x;
	
	for(int dest_y = gind_y; dest_y < y_size; dest_y += y_threads)
	{
		for(int dest_x = gind_x; dest_x < x_size; dest_x += x_threads)
		{
			float h_0 = raytrace_maxheight_f(source_x, source_y, dest_x, dest_y, y_size, height_map);
			
			int flat_idx	= dest_x * y_size + dest_y;		//I think it's column major
			float h_2	= height_map[flat_idx];
			
			Height_Results results = raytrace_maxheight_f(source_x, source_y, dest_x, dest_y, y_size, height_map);
			
			float x_t_o	= (float)(results.ind_x - source_x) * .1;	//100 meters per pixel. distance in kilometers
			float y_t_o	= (float)(results.ind_y - source_y) * .1;
			float d_1	= sqrtf((x_t_o * x_t_o) + (y_t_o * y_t_o));
			
			float x_o_r	= (float)(dest_x - results.ind_x) * .1;	//100 meters per pixel. distance in kilometers
			float y_o_r	= (float)(dest_y - results.ind_y) * .1;
			float d_2	= sqrtf((x_o_r * x_o_r) + (y_o_r * y_o_r));
			
			float loss	= calc_loss(h_0, h_1, h_2, d_1, d_2, freq);
			pLoss[i]	= loss  
		}
	} 
}

extern "C" 
{
//source_z should be the abosolute height of the emmiter, not it's height over the ground
void signal_loss(const int source_x, const int source_y, const int source_z, const float *height_map, float *pLoss, const int y_size, const int x_size)
{
	size_t mem_size = y_size * x_size * sizeof(float);
	float *height_map_dev, *pLoss_dev;
	
	hipError_t status;
	status = hipMalloc((void**)&height_map_dev, mem_size);
	status = hipMalloc((void**)&pLoss_dev, mem_size);
	
	status = hipMemcpy(height_map_dev, height_map, mem_size, hipMemcpyHostToDevice); 
	status = hipMemcpy(pLoss_dev, pLoss, mem_size, hipMemcpyHostToDevice); 
	
	// -- Skipping multi-gpu or multi-stream for this --
	
	int deviceID;								//get device ID
	hipGetDevice(&deviceID);
	hipDeviceProp_t props;							//get device properties
	hipGetDeviceProperties((void**)&props, deviceID);
	unsigned ThreadsPerBlock	= props.warpSize * 4;			//threads per block should be soe multiple of warpsize or just set to props.maxThreadsPerBlock
	unsigned BlocksPerGrid		= props.multiProcessorCount * 2;	
	
	calc_losses<<<BlocksPerGrid, ThreadsPerBlock>>>
	
	hipDeviceSynchronize();
	
	hipFree(pLoss_dev);
	hipFree(height_map_dev);
}

}
