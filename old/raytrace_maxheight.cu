#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

extern "C" 
{
__host__ __device__ 
float raytrace_maxheight_f(const int source_x, const int source_y, const int dest_x, const int dest_y, const int map_y_size, const float *height_map)
{
	const int x_signed_len = dest_x - source_x;
	const int y_signed_len = dest_y - source_y;
	const int length = max(abs(x_signed_len), abs(y_signed_len));
	float max_height = -99999.0;
	
	for(int offset = 0; offset < (length + 1); offset++)			//length + 1 to include endpoint
	{
		float fraction = (float)offset / (float)length;
		int x = (int)roundf(x_signed_len * fraction + source_x);
		int y = (int)roundf(y_signed_len * fraction + source_y);
		int i = x * map_y_size + y;					//seems to be column major
		float height = height_map[i];
		//printf("c height for %i, %i, : %f\n", x, y, height);
		if(height > max_height)
			max_height = height;
	}
	
	return max_height;
}

float accept(const int bent, const int tint, const int gent, const int went, const float *height_map)
{
	return height_map[0];
}
}

